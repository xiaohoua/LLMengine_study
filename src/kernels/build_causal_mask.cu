#include "hip/hip_runtime.h"
#include "src/kernels/build_casual_mask.h"

template<typename T>
__global__ void BuildCausalMasksConsideringContextPastKV(T* mask,
                                                const int* q_lens,  //input lens, shape=[batch size]
                                                const int* k_lens,  //context lens, shape=[batch size]
                                                int max_q_len,  // max(q_lens)
                                                int max_k_len){ // max(k_lens)
                                                
    int tid = threadIdx.x;
    int qlen = q_lens[blockIdx.x];
    int klen = k_lens[blockIdx.x];

    mask += blockIdx.x * max_q_len*max_k_len;

    while(tid < max_q_len*max_k_len){
        int q = tid / max_k_len;
        int k = tid % max_k_len;

        bool is_one = q < qlen && k < klen && k <= q + (klen - qlen) && k >= klen - qlen;
        mask[tid] = static_cast<T>(is_one);

        // 保证遍历完一个bs中所有的空间
        tid += blockDim.x;
    }
}

template<typename T>
void launchBuildCausalMasks(TensorWrapper<T>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens)
{
    int batch_size = mask->shape[0];
    int max_q_len = mask->shape[1];
    int max_k_len = mask->shape[2];
    // XuLin-1017: 此处的max_q_len和max_k_len是经过统计后得出的外部输入
    BuildCausalMasksConsideringContextPastKV<T><<<batch_size, 256>>>(mask->data, q_lens->data, k_lens->data, max_q_len, max_k_len);
}

template void launchBuildCausalMasks(TensorWrapper<float>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens);

template void launchBuildCausalMasks(TensorWrapper<half>* mask, 
                            TensorWrapper<int>* q_lens, 
                            TensorWrapper<int>* k_lens);